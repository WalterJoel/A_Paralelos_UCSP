#include <stdio.h>
#include <hip/hip_runtime.h>
#include <time.h>


__global__
void kernel(float *vec, float *mat, float *out, const int N, const int M)
{
    int tid=threadIdx.x+blockIdx.x*blockDim.x;
    float sum=0;
    if(tid<M)
    {
        for(int i=0; i<N; i++)
            out[tid] += vec[i]*mat[(i*M)+tid];
    }
}

void init_array(float *a, const int N);
void init_mat(float *a, const int N, const int M);
void print_array(float *a, const int N, char *d);
void print_mat(float *a, const int N, const int M, char *d);

int main (void) 
{
	srand( time(NULL) );

    float *a, *b, *c;
    float *dev_a, *dev_b, *dev_c;

    int N=5;
    int M=7;
    a=(float*)malloc(sizeof(float)*N);
    b=(float*)malloc(sizeof(float)*N*M);
    c=(float*)malloc(sizeof(float)*M);
    init_array(a, N);
    init_mat(b, N, M);
    init_array(c, M);

    printf("<<<<<<<<<< initial data:\n");
    print_array(a, N, "in-vector");
    print_mat(b, N, M, "matrix");
    print_array(c, M, "out-vector");

    hipMalloc((void**)&dev_a, sizeof(float)*N);
    hipMalloc((void**)&dev_b, sizeof(float)*N*M);
    hipMalloc((void**)&dev_c, sizeof(float)*M);

    hipMemcpy(dev_a, a, sizeof(float)*N, hipMemcpyHostToDevice);
    hipMemcpy(dev_b, b, sizeof(float)*N*M, hipMemcpyHostToDevice);

    kernel<<<M/256+1, 256>>>(dev_a, dev_b, dev_c, N, M);
    //printf("error code: %s\n",cudaGetErrorString(cudaGetLastError()));

    hipMemcpy(c, dev_c, sizeof(float)*M, hipMemcpyDeviceToHost);

    hipFree(dev_a);
    hipFree(dev_b);
    hipFree(dev_c);

    printf(">>>>>>>>>> RESULTADPS:\n");
    print_array(c, M, "out-vector");

    return 0;
};

void init_array(float *a, const int N) 
{
    int i;
    for(i=0; i<N; i++)
        a[i] = rand() % 4 + 1;
}
void init_mat(float *a, const int N, const int M) 
{
    int i, j;
    for(i=0; i<N; i++)
        for(j=0; j<M; j++)
            a[i*M+j] = rand() % 4 + 1;
}
void print_array(float *a, const int N, char *d) {
    int i;
    for(i=0; i<N; i++)
        printf("\n%s[%d]: %f",d, i, a[i]);
    printf("\n");
}
void print_mat(float *a, const int N, const int M, char *d) {
    int i, j;
    for(i=0; i<N; i++){
	    printf("\n%s[%d]:", d, i);
    	for (j=0; j<M; j++)
            printf("\t%6.4f", a[i*M+j]);
    }
    printf("\n");
}